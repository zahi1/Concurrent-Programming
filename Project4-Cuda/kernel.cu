#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstring>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>
#include ""
#include <sstream>

// Define Student structure
struct Student {
    char name[256];
    int year;
    float grade;
};

// Define Result structure
struct Result {
    char data[256];
};

__device__ char gradeFromScore(float score) {
    if (score >= 9) return 'A';
    else if (score >= 8) return 'B';
    else if (score >= 7) return 'C';
    else if (score >= 6) return 'D';
    else return 'F';
}

__global__ void processStudents(Student* students, Result* results, int numStudents, int* resultCounter) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numStudents) {
        Student s = students[idx];
        if (s.name[0] > 'P') {
            // Convert name to uppercase
            for (int i = 0; s.name[i] != '\0'; i++) {
                if (s.name[i] >= 'a' && s.name[i] <= 'z') {
                    s.name[i] = s.name[i] - 'a' + 'A';
                }
            }

            // Prepare result string
            char resultString[256];
            int cursor = 0;
            for (int i = 0; s.name[i] != '\0'; i++) {
                resultString[cursor++] = s.name[i];
            }
            resultString[cursor++] = '-';

            // Convert year to string and append
            int year = s.year;

            resultString[cursor++] = '0' + year;

            // Append grade
            char grade = gradeFromScore(s.grade);
            resultString[cursor++] = grade;
            resultString[cursor] = '\0';  // Null-terminate the string

            // Write to result array using atomic operation
            int pos = atomicAdd(resultCounter, 1);
            for (int i = 0; resultString[i] != '\0'; i++) {
                results[pos].data[i] = resultString[i];
            }
            results[pos].data[cursor] = '\0'; // Null-terminate the result
        }
    }
}


std::vector<Student> readStudentsFromFile(const char* filename) {
    std::vector<Student> students;
    std::ifstream file(filename);
    std::string line;

    while (std::getline(file, line)) {
        Student s;
        sscanf(line.c_str(), "%[^,],%d,%f", s.name, &s.year, &s.grade);
        students.push_back(s);
    }

    file.close();
    return students;
}

int main() {
    // Read students from file
    std::vector<Student> studentVector = readStudentsFromFile("data1.txt");
    int numStudents = studentVector.size();

    // Prepare arrays for CUDA
    Student* h_students = new Student[numStudents];
    Result* h_results = new Result[numStudents];
    int h_resultCounter = 0;

    for (int i = 0; i < numStudents; ++i) {
        h_students[i] = studentVector[i];
    }

    // Allocate memory on GPU
    Student* d_students;
    Result* d_results;
    int* d_resultCounter;
    hipMalloc(&d_students, numStudents * sizeof(Student));
    hipMalloc(&d_results, numStudents * sizeof(Result));
    hipMalloc(&d_resultCounter, sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_students, h_students, numStudents * sizeof(Student), hipMemcpyHostToDevice);
    hipMemcpy(d_resultCounter, &h_resultCounter, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 64; // or another appropriate value
    int numBlocks = (numStudents + blockSize - 1) / blockSize;
    processStudents << <numBlocks, blockSize >> > (d_students, d_results, numStudents, d_resultCounter);


    // Copy results back to host
    hipMemcpy(h_results, d_results, numStudents * sizeof(Result), hipMemcpyDeviceToHost);
    hipMemcpy(&h_resultCounter, d_resultCounter, sizeof(int), hipMemcpyDeviceToHost);

    // Write results to a file
    std::ofstream outFile("result.txt");
    for (int i = 0; i < h_resultCounter; i++) {
        outFile << h_results[i].data << std::endl;
    }
    outFile.close();

    // Free memory
    delete[] h_students;
    delete[] h_results;
    hipFree(d_students);
    hipFree(d_results);
    hipFree(d_resultCounter);

    return 0;
}